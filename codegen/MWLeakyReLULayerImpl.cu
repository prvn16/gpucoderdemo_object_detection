#include "hip/hip_runtime.h"
#include "MWLeakyReLULayerImpl.hpp"
 void __global__ leakyReLUImpl(float * AFQBkxwYGKLsACiDKwRM, const double AwZQzUhuWVLGrWgLHRuM, 
const int CpMjJjtGOeWOzwxpAAQP) { int const i = blockDim.x * blockIdx.x + threadIdx.x; 
if (i < CpMjJjtGOeWOzwxpAAQP) { float tf = float(AFQBkxwYGKLsACiDKwRM[i]<0); AFQBkxwYGKLsACiDKwRM[i] = 
AFQBkxwYGKLsACiDKwRM[i] - tf*AwZQzUhuWVLGrWgLHRuM*AFQBkxwYGKLsACiDKwRM[i]; } } void 
leakyReLUForwardImpl(int YgcpEBUCwCLaPhyntIio, int vIWQzNvYZSuxmOTVDFhU, int 
gzSTokDHvkXefhiGDcWL, int LtEgcYoEYjkrWuohutgw,  const double 
kkqTyvjYvRFtTOyQUwrF, float* output) { int fjfzkUfcCOqjrkAVGfuc = 
YgcpEBUCwCLaPhyntIio*vIWQzNvYZSuxmOTVDFhU* 
gzSTokDHvkXefhiGDcWL*LtEgcYoEYjkrWuohutgw; int 
pckLLTEdVPoCZLRwyDnM = (fjfzkUfcCOqjrkAVGfuc < 1024) ? fjfzkUfcCOqjrkAVGfuc : 
1024; int MEmIeGILUZNEWEagSzRk = (fjfzkUfcCOqjrkAVGfuc + 
pckLLTEdVPoCZLRwyDnM - 1)/pckLLTEdVPoCZLRwyDnM; 
leakyReLUImpl<<<MEmIeGILUZNEWEagSzRk, pckLLTEdVPoCZLRwyDnM>>>( 
output, (1 - kkqTyvjYvRFtTOyQUwrF), fjfzkUfcCOqjrkAVGfuc); }