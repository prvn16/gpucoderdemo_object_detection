#include "MWLeakyReLULayer.hpp"
#include "MWLeakyReLULayerImpl.hpp"
 MWLeakyReLULayer::MWLeakyReLULayer() { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWLeakyReLULayer::~MWLeakyReLULayer() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); } void 
MWLeakyReLULayer::createLeakyReLULayer(MWCNNLayer* jaqKGCwoANNDMHgAsehk, 
double JwxFdqOKggeawILBfGgg) {  setTop(jaqKGCwoANNDMHgAsehk); 
setBatchSize(getTop()->getBatchSize()); setHeight(getTop()->getHeight()); 
setWidth(getTop()->getWidth()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); kkqTyvjYvRFtTOyQUwrF = 
JwxFdqOKggeawILBfGgg; 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, getBatchSize(), getNumOutputFeatures(), 
getHeight(), getWidth())); setData(getTop()->getData()); return; } void 
MWLeakyReLULayer::predict() { leakyReLUForwardImpl(getHeight(), getWidth(), 
getNumInputFeatures(), getBatchSize(), kkqTyvjYvRFtTOyQUwrF, getData());  return; } 
void MWLeakyReLULayer::cleanup() { return; }